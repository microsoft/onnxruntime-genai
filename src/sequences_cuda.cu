#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "span.h"

namespace Generators {
namespace cuda {

__global__ void ExpandInputSequences(const int32_t* input_sequences, int32_t* sequences, int batch_size, int beam_size, int current_length, int max_length) {
  // The original inputs are not expanded, this expands them in place into the sequences
  for (size_t batch = 0; batch < batch_size; batch++) {
    for (size_t beam = 0; beam < beam_size; beam++) {
      for (int j = 0; j < current_length; j++) {
        sequences[(batch * beam_size + beam) * max_length + j] =
            static_cast<int32_t>(input_sequences[batch * current_length + j]);
      }
    }
  }
}

void Launch_ExpandInputSequences(std::span<const int32_t> input_sequences, std::span<int32_t> sequences, int batch_size, int beam_size, int current_length, int max_length, hipStream_t stream) {
  ExpandInputSequences<<<1, 1, 0, stream>>>(input_sequences.data(), sequences.data(), batch_size, beam_size, current_length, max_length);
}

__global__ void AppendNextTokenToSequences(const int32_t* next_tokens, int32_t* sequences, int batch_beam_size, int current_length, int max_length) {
  // Append next token to each sequence.
  for (int i = 0; i < batch_beam_size; i++) {
    sequences[i * max_length + current_length] = next_tokens[i];
  }
}

void Launch_AppendNextTokenToSequences(std::span<const int32_t> next_tokens, std::span<int32_t> sequences, int batch_beam_size, int current_length, int max_length, hipStream_t stream) {
  AppendNextTokenToSequences<<<1, 1, 0, stream>>>(next_tokens.data(), sequences.data(), batch_beam_size, current_length, max_length);
}

}  // namespace cuda
}  // namespace Generators
