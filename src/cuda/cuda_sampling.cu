#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include <math.h>
#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <limits>
#include <stdio.h>
#include "cuda_sampling.h"
#include "cuda_topk.h"
#include "cuda_topk_softmax.cuh"
#include "smartptrs.h"
#include "span.h"

namespace Generators {
namespace cuda {

// Initializes the cuRAND states for each batch item.
__global__ void InitCurandStates(unsigned long long seed, hiprandState* states, int batch_size) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= batch_size) return;
  hiprand_init(seed, index, 0, &states[index]);
}

void SamplingData::ReInitCurandStates(unsigned long long random_seed, int batch_size, hipStream_t stream) {
  random_seed_ = random_seed;
  InitCurandStates<<<CeilDiv(batch_size, 128), 128, 0, stream>>>(random_seed, curand_states, batch_size);
  CUDA_CHECK_LAUNCH();
}

size_t SamplingData::CalculateTotalSize(int batch_size, int vocab_size, hipStream_t stream) {
  // Get size from base class and add our own buffer sizes
  size_t total_size = TopkData::CalculateTotalSize(batch_size, vocab_size, stream);
  size_t vocab_batch_size = static_cast<size_t>(vocab_size) * batch_size;

  total_size += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);  // prefix_sums
  total_size += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);  // scores_adjusted
  total_size += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);  // prefix_sums_adjusted
  total_size += AlignUp(batch_size * sizeof(float), kGpuBufferAlignment);        // thresholds
  total_size += AlignUp(batch_size * sizeof(hiprandState), kGpuBufferAlignment);  // curand_states

  return total_size;
}

void SamplingData::InitializeBuffers(int batch_size, int vocab_size, hipStream_t stream) {
  // The base class constructor has already called TopkData::InitializeBuffers.
  // This override is only responsible for initializing the members of the derived class.

  // Calculate the starting offset for the derived class members by getting the total size of the base class.
  uint8_t* current_ptr = memory_buffer_span_.data() + TopkData::CalculateTotalSize(batch_size, vocab_size, stream);

  size_t vocab_batch_size = static_cast<size_t>(vocab_size) * batch_size;

  prefix_sums = reinterpret_cast<float*>(current_ptr);
  current_ptr += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);

  scores_adjusted = reinterpret_cast<float*>(current_ptr);
  current_ptr += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);

  prefix_sums_adjusted = reinterpret_cast<float*>(current_ptr);
  current_ptr += AlignUp(vocab_batch_size * sizeof(float), kGpuBufferAlignment);

  thresholds = reinterpret_cast<float*>(current_ptr);
  current_ptr += AlignUp(batch_size * sizeof(float), kGpuBufferAlignment);

  curand_states = reinterpret_cast<hiprandState*>(current_ptr);
}

SamplingData::SamplingData(unsigned long long random_seed, int batch_size, int vocab_size, hipStream_t stream, void* buffer, size_t buffer_size)
    : TopkData(batch_size, vocab_size, stream, buffer,
               buffer ? buffer_size : CalculateTotalSize(batch_size, vocab_size, stream)) {
  // The base constructor handles buffer allocation/wrapping.
  // We just need to re-initialize the pointers for the derived class members.
  InitializeBuffers(batch_size, vocab_size, stream);
  ReInitCurandStates(random_seed, batch_size, stream);
}

// A fused kernel that performs all steps of Top-P sampling on a pre-selected set of Top-K candidates.
// This monolithic approach minimizes kernel launch overhead and maximizes data locality by using shared memory.
// It has been empirically shown to be the most performant approach for k <= 256.
template <int kBlockSize>
__global__ void FusedSamplingKernel(int32_t* next_token_out, const float* scores, const int* indices, int k,
                                    float p, float temperature, int stride, hiprandState* curand_states) {
  const int batch_idx = blockIdx.x;
  const float* batch_scores = scores + batch_idx * stride;
  const int* batch_indices = indices + batch_idx * stride;

  // Allocate shared memory for all intermediate data. This is the key to performance.
  extern __shared__ float smem[];
  float* temp_scaled_logits = smem;
  float* filtered_logits = smem + kBlockSize;

  typedef hipcub::BlockReduce<float, kBlockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage reduce_temp_storage;
  __shared__ float block_max_val;
  __shared__ float block_sum_exp;

  // --- Stage 1: Initial Softmax with Temperature (for Top-P filtering) ---

  // Apply temperature scaling.
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    temp_scaled_logits[i] = batch_scores[i] / temperature;
  }

  // For sorted input, the max score is always the first element.
  if (threadIdx.x == 0) {
    block_max_val = batch_scores[0] / temperature;
  }
  __syncthreads();

  float thread_val = 0.0f;
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    thread_val += expf(temp_scaled_logits[i] - block_max_val);
  }
  float reduced_sum = BlockReduce(reduce_temp_storage).Reduce(thread_val, hipcub::Sum());
  if (threadIdx.x == 0) block_sum_exp = reduced_sum;
  __syncthreads();

  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    temp_scaled_logits[i] =
        (block_sum_exp > 0.0f) ? (expf(temp_scaled_logits[i] - block_max_val) / block_sum_exp) : 0.0f;
  }
  __syncthreads();

  // --- Stage 2: Compute Initial CDF (in-place scan on initial probabilities) ---
  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage scan_temp_storage;
  float running_total = 0.0f;
  for (int i = 0; i < k; i += kBlockSize) {
    float score = (threadIdx.x + i < k) ? temp_scaled_logits[threadIdx.x + i] : 0.0f;
    float scanned_score;
    BlockScan(scan_temp_storage).InclusiveSum(score, scanned_score);
    __syncthreads();

    if (threadIdx.x + i < k) temp_scaled_logits[threadIdx.x + i] = scanned_score + running_total;
    __syncthreads();

    if (threadIdx.x == kBlockSize - 1) running_total += scanned_score;
    __syncthreads();
  }

  // --- Stage 3: Filter SCALED logits based on the CDF ---
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    const float prev_sum = (i == 0) ? 0.0f : temp_scaled_logits[i - 1];
    // Reread scaled logits from global memory to filter
    float current_scaled_logit = batch_scores[i] / temperature;
    filtered_logits[i] = (prev_sum < p) ? current_scaled_logit : -FLT_MAX;
  }
  __syncthreads();

  // --- Stage 4: Re-normalize filtered logits (temperature=1.0 as it's already baked in) ---
  thread_val = -FLT_MAX;
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    thread_val = max(thread_val, filtered_logits[i]);
  }
  float reduced_max = BlockReduce(reduce_temp_storage).Reduce(thread_val, hipcub::Max());
  if (threadIdx.x == 0) block_max_val = reduced_max;
  __syncthreads();

  thread_val = 0.0f;
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    thread_val += expf(filtered_logits[i] - block_max_val);
  }
  reduced_sum = BlockReduce(reduce_temp_storage).Reduce(thread_val, hipcub::Sum());
  if (threadIdx.x == 0) block_sum_exp = reduced_sum;
  __syncthreads();

  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    filtered_logits[i] = (block_sum_exp > 0.0f) ? (expf(filtered_logits[i] - block_max_val) / block_sum_exp) : 0.0f;
  }
  __syncthreads();

  // --- Stage 5: Compute Final CDF (in-place scan on final probabilities) ---
  running_total = 0.0f;
  for (int i = 0; i < k; i += kBlockSize) {
    float score = (threadIdx.x + i < k) ? filtered_logits[threadIdx.x + i] : 0.0f;
    float scanned_score;
    BlockScan(scan_temp_storage).InclusiveSum(score, scanned_score);
    __syncthreads();
    if (threadIdx.x + i < k) filtered_logits[threadIdx.x + i] = scanned_score + running_total;
    __syncthreads();
    if (threadIdx.x == kBlockSize - 1) running_total += scanned_score;
    __syncthreads();
  }

  // --- Stage 6 & 7: Sample via Parallel Search ---
  __shared__ int selected_index_smem;
  __shared__ float threshold_smem;

  if (threadIdx.x == 0) {
    // Use min to prevent multiplying down the random value, which could introduce bias.
    // This robustly handles the case where hiprand_uniform is exactly 1.0.
    threshold_smem = min(hiprand_uniform(&curand_states[batch_idx]), 0.9999999f);
    selected_index_smem = k - 1;
  }
  __syncthreads();

  // All threads in the block search in parallel for the first index that meets the threshold.
  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    if (filtered_logits[i] >= threshold_smem) {
      atomicMin(&selected_index_smem, i);
      break;  // Early exit for this thread
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    next_token_out[batch_idx] = batch_indices[selected_index_smem];
  }
}

// Kernels for the multi-stage sampling pipeline (used for k > 256)
#pragma region MultiStageKernels

template <int kBlockSize>
__global__ void CorrectPrefixSumKernel(const float* scores, float* prefix_sums, int k) {
  const int batch_idx = blockIdx.x;
  const float* batch_scores = scores + batch_idx * k;
  float* batch_prefix_sums = prefix_sums + batch_idx * k;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ float chunk_total;
  float running_total = 0.0f;

  for (int i = 0; i < k; i += kBlockSize) {
    float score = (threadIdx.x + i < k) ? batch_scores[threadIdx.x + i] : 0.0f;
    float scanned_score;
    BlockScan(temp_storage).InclusiveSum(score, scanned_score);
    __syncthreads();
    if (threadIdx.x + i < k) {
      batch_prefix_sums[threadIdx.x + i] = scanned_score + running_total;
    }
    __syncthreads();
    if (threadIdx.x == kBlockSize - 1) {
      chunk_total = scanned_score;
    }
    __syncthreads();
    running_total += chunk_total;
  }
}

__global__ void FilterOnTopPKernel(float* filtered_logits, const float* original_logits, const float* cdf, int k,
                                   float p, float temperature, int stride) {
  const int batch_idx = blockIdx.x;
  for (int i = threadIdx.x; i < k; i += blockDim.x) {
    const float prev_sum = (i == 0) ? 0.0f : cdf[batch_idx * k + i - 1];
    float scaled_logit = original_logits[batch_idx * stride + i] / temperature;
    filtered_logits[batch_idx * k + i] = (prev_sum < p) ? scaled_logit : -FLT_MAX;
  }
}

__global__ void RandomThresholdKernel(hiprandState* curand_states, float* thresholds, int batch_size) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < batch_size) {
    // Use min to prevent multiplying down the random value, which could introduce bias.
    thresholds[i] = min(hiprand_uniform(&curand_states[i]), 0.9999999f);
  }
}

template <int kBlockSize>
__global__ void SampleKernel(int32_t* next_token_out, const int* indices, const float* cdf, int k, int stride,
                             const float* thresholds) {
  const int batch_idx = blockIdx.x;
  const float threshold = thresholds[batch_idx];
  __shared__ int selected_index_smem;

  if (threadIdx.x == 0) selected_index_smem = k - 1;
  __syncthreads();

  for (int i = threadIdx.x; i < k; i += kBlockSize) {
    if (cdf[batch_idx * k + i] >= threshold) {
      atomicMin(&selected_index_smem, i);
      break;
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    next_token_out[batch_idx] = indices[batch_idx * stride + selected_index_smem];
  }
}
#pragma endregion

// A multi-stage sampling pipeline that is more efficient for large k.
void LaunchMultiStageSampleKernel(SamplingData* data, hipStream_t stream, const float* scores, const int* indices,
                                  int32_t* next_token_out, int k, int batch_size, float p, float temperature,
                                  int stride) {
  dim3 grid(batch_size);
  dim3 block(256);

  // Stage 1: Initial Softmax with Temperature.
  ApplySoftmaxToSortedTopK<false>(stream, data->prefix_sums_adjusted, nullptr, scores, nullptr, k, batch_size,
                                  stride, temperature);

  // Stage 2: Compute Initial CDF.
  CorrectPrefixSumKernel<256><<<grid, block, 0, stream>>>(data->prefix_sums_adjusted, data->prefix_sums, k);

  // Stage 3: Filter scaled logits.
  FilterOnTopPKernel<<<grid, block, 0, stream>>>(data->scores_adjusted, scores, data->prefix_sums, k, p,
                                                 temperature, stride);

  // Stage 4: Re-normalize filtered logits (temperature is already baked in).
  ApplySoftmaxToSortedTopK<false>(stream, data->prefix_sums_adjusted, nullptr, data->scores_adjusted,
                                  nullptr, k, batch_size, k, 1.0f);

  // Stage 5: Compute Final CDF.
  CorrectPrefixSumKernel<256><<<grid, block, 0, stream>>>(data->prefix_sums_adjusted, data->prefix_sums, k);

  // Stage 6: Generate random thresholds.
  RandomThresholdKernel<<<CeilDiv(batch_size, 256), block, 0, stream>>>(data->curand_states,
                                                                        data->thresholds, batch_size);

  // Stage 7: Sample via Parallel Search.
  SampleKernel<256><<<grid, block, 0, stream>>>(next_token_out, indices, data->prefix_sums, k, stride,
                                                data->thresholds);
}

void LaunchFusedSampleKernel(SamplingData* data, hipStream_t stream, const float* scores, const int* indices,
                             int32_t* next_token_out, int k, int batch_size, float p, float temperature, int stride) {
  assert(k <= kFusedSamplingMaxK);
  dim3 grid(batch_size);
  constexpr int block_size = 256;
  dim3 block(block_size);

  // Shared memory size is determined by the needs of the fused kernel: two float arrays of size block_size.
  constexpr size_t shared_mem_bytes = 2 * block_size * sizeof(float);

  FusedSamplingKernel<block_size><<<grid, block, shared_mem_bytes, stream>>>(
      next_token_out, scores, indices, k, p, temperature, stride, data->curand_states);
}

void GetSample(SamplingData* data, hipStream_t stream, int32_t* next_token_out, const float* scores_in,
               int vocab_size, int batch_size, int k, float p, float temperature) {
  if (k <= 0 || k > vocab_size) {
    k = vocab_size;
  }

  RunTopK(data, stream, scores_in, vocab_size, batch_size, k);
  const float* topk_scores = data->topk_scores;
  const int* topk_indices = data->topk_indices;
  int topk_stride = data->topk_stride;

  // The fused kernel is the most performant approach for k up to 256.
  if (k <= kFusedSamplingMaxK) {
    LaunchFusedSampleKernel(data, stream, topk_scores, topk_indices, next_token_out, k, batch_size, p,
                            temperature, topk_stride);
  } else {
    // Fall back to multi-stage sampling pipeline. This is not a typical use case.
    LaunchMultiStageSampleKernel(data, stream, topk_scores, topk_indices, next_token_out, k, batch_size, p,
                                 temperature, topk_stride);
  }
  CUDA_CHECK_LAUNCH();
}

// Implementation for the general-purpose block-wise softmax, used by beam search.
template <int kBlockSize, bool is_log_softmax>
__global__ void BlockwiseSoftmaxKernel(float* output, const float* input, int softmax_elements, int input_stride,
                                       int output_stride) {
  const int batch_idx = blockIdx.x;
  const float* batch_input = input + batch_idx * input_stride;
  float* batch_output = output + batch_idx * output_stride;

  typedef hipcub::BlockReduce<float, kBlockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ float max_val;
  __shared__ float sum_exp;

  // Step 1: Find max value in parallel for numerical stability.
  float thread_max = -std::numeric_limits<float>::max();
  for (int i = threadIdx.x; i < softmax_elements; i += kBlockSize) {
    thread_max = max(thread_max, batch_input[i]);
  }
  float block_max = BlockReduce(temp_storage).Reduce(thread_max, hipcub::Max());
  if (threadIdx.x == 0) {
    max_val = block_max;
  }
  __syncthreads();

  // Step 2: Compute sum of exponents in parallel.
  float thread_sum_exp = 0.0f;
  for (int i = threadIdx.x; i < softmax_elements; i += kBlockSize) {
    thread_sum_exp += expf(batch_input[i] - max_val);
  }
  float block_sum = BlockReduce(temp_storage).Reduce(thread_sum_exp, hipcub::Sum());
  if (threadIdx.x == 0) {
    sum_exp = block_sum;
  }
  __syncthreads();

  // Step 3: Compute final softmax or log_softmax and write to output.
  if constexpr (is_log_softmax) {
    // Add a small epsilon to prevent log(0) which results in -inf.
    float log_sum_exp = logf(sum_exp + 1e-20f);
    for (int i = threadIdx.x; i < softmax_elements; i += kBlockSize) {
      batch_output[i] = batch_input[i] - max_val - log_sum_exp;
    }
  } else {
    for (int i = threadIdx.x; i < softmax_elements; i += kBlockSize) {
      // Handle case where sum_exp is zero to prevent division by zero (NaN).
      batch_output[i] = (sum_exp > 0.0f) ? (expf(batch_input[i] - max_val) / sum_exp) : 0.0f;
    }
  }
}

template <bool is_log_softmax>
void DispatchBlockwiseSoftmaxForward(hipStream_t stream, float* output, const float* input, int softmax_elements,
                                     int input_stride, int output_stride, int batch_count) {
  // This kernel is efficient for large softmax_elements (like vocab_size) where
  // a single block can cooperatively process one batch item.
  constexpr int kBlockSize = 256;
  dim3 grid(batch_count);
  dim3 block(kBlockSize);

  BlockwiseSoftmaxKernel<kBlockSize, is_log_softmax><<<grid, block, 0, stream>>>(output, input, softmax_elements,
                                                                                 input_stride, output_stride);
  CUDA_CHECK_LAUNCH();
}

// Explicitly instantiate the templates to be linked from other translation units.
template void DispatchBlockwiseSoftmaxForward<true>(hipStream_t, float*, const float*, int, int, int, int);
template void DispatchBlockwiseSoftmaxForward<false>(hipStream_t, float*, const float*, int, int, int, int);

}  // namespace cuda
}  // namespace Generators
