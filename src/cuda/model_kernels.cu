// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits>
#include <assert.h>
#include <stdio.h>

namespace Generators {
namespace cuda {

template <typename T>
__global__ void UpdatePositionIds(T* positions, int batch_beam_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_beam_size)
    positions[i]++;
}

template <typename T>
void Launch_UpdatePositionIds(T* positions, int batch_beam_size, hipStream_t stream) {
  UpdatePositionIds<T><<<(batch_beam_size + 255) / 256, 256, 0, stream>>>(positions, batch_beam_size);
}

template void Launch_UpdatePositionIds(int32_t* positions, int batch_beam_size, hipStream_t stream);
template void Launch_UpdatePositionIds(int64_t* positions, int batch_beam_size, hipStream_t stream);

template <typename T>
__global__ void CopyAndUpdateAttentionMask(T* mask_data, const T* old_mask_data, int batch_beam_size,
                                           int current_length, int max_length) {
  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  int i = global_index / current_length;
  int j = global_index % current_length;
  if (i < batch_beam_size) {
    if (j < current_length - 1) {
      mask_data[i * max_length + j] = old_mask_data[i * (current_length - 1) + j];
    } else {
      mask_data[i * max_length + j] = 1;
    }
  }
}

template <typename T>
__global__ void UpdateAttentionMask(T* mask_data, int batch_beam_size, int current_length, int max_length) {
  int i = blockIdx.x;
  if (i < batch_beam_size) {
    mask_data[i * max_length + current_length] = 1;
  }
}

template <typename T>
void Launch_UpdateAttentionMask(T* mask_data, const T* old_mask_data, int batch_beam_size, int current_length,
                                int max_length, bool update_only, hipStream_t stream) {
  if (update_only) {
    UpdateAttentionMask<T>
        <<<batch_beam_size, 1, 0, stream>>>(mask_data, batch_beam_size, current_length, max_length);
  } else {
    CopyAndUpdateAttentionMask<T><<<(batch_beam_size * max_length + 255) / 256, 256, 0, stream>>>(
        mask_data, old_mask_data, batch_beam_size, current_length, max_length);
  }
}

template void Launch_UpdateAttentionMask(int32_t* mask_data, const int32_t* old_mask_data, int batch_beam_size,
                                         int current_length, int max_length, bool update_only, hipStream_t stream);
template void Launch_UpdateAttentionMask(int64_t* mask_data, const int64_t* old_mask_data, int batch_beam_size,
                                         int current_length, int max_length, bool update_only, hipStream_t stream);

__global__ void HandleEOSArray(float* batch_logits, int batch_beam_size, int vocab_size, const int32_t* eos_token_ids, int eos_token_ids_count) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= batch_beam_size)
    return;

  float* logits = batch_logits + index * vocab_size;
  float max = std::numeric_limits<float>::lowest();
  for (int i = 0; i < eos_token_ids_count; i++) {
    max = std::max(max, logits[eos_token_ids[i]]);
    logits[eos_token_ids[i]] = std::numeric_limits<float>::lowest();  // Set all EOS token options to never happen (the first will get the max of all)
  }

  logits[eos_token_ids[0]] = max;  // Set the score of the primary EOS token to the highest of any of the EOS tokens
}

void LaunchHandleEOSArray(float* batch_logits, int batch_beam_size, int vocab_size, const int32_t* eos_token_ids, int eos_token_ids_count, hipStream_t stream) {
  HandleEOSArray<<<(batch_beam_size + 255) / 256, 256, 0, stream>>>(batch_logits, batch_beam_size, vocab_size, eos_token_ids, eos_token_ids_count);
}

__global__ void ConvertFp16ToFp32(const half* src, float* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __half2float(src[idx]);
}

void LaunchFp16ToFp32(const uint16_t* fp16, float* fp32, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp16ToFp32<<<num_blocks, block_size, 0, stream>>>(reinterpret_cast<const half*>(fp16), fp32, count);
}

__global__ void ConvertFp32ToFp16(const float* src, half* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    dst[idx] = __float2half(src[idx]);
}

void LaunchFp32ToFp16(const float* fp32, uint16_t* fp16, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertFp32ToFp16<<<num_blocks, block_size, 0, stream>>>(fp32, reinterpret_cast<half*>(fp16), count);
}

__global__ void ConvertInt32ToInt64(const int32_t* src, int64_t* dst, int count) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    dst[idx] = src[idx];
  }
}

void LaunchInt32ToInt64(const int32_t* src, int64_t* dst, int count, hipStream_t stream) {
  int block_size = 256;
  int num_blocks = (count + block_size - 1) / block_size;
  ConvertInt32ToInt64<<<num_blocks, block_size, 0, stream>>>(src, dst, count);
}

namespace {

struct ReorderPastStateParams {
  // Support head_size up to 128
  constexpr static unsigned int kTileSize = 32;
  constexpr static unsigned int kSeqTileSize = 16;
};

}  // namespace

__global__ void ReorderPastStatesKernel(float4* out_buffer,
                                        const float4* in_buffer,
                                        int batch_size,
                                        int num_heads,
                                        int max_length,
                                        int chunked_head_size) {
  __shared__ float4 tile[ReorderPastStateParams::kSeqTileSize][ReorderPastStateParams::kTileSize + 1];

  const int b = blockIdx.z;
  const int n = blockIdx.y;
  const int s_base = blockIdx.x * ReorderPastStateParams::kSeqTileSize;
  const int s = s_base + threadIdx.y;
  const int base_offset = (b * num_heads + n) * max_length * chunked_head_size;

  if (s < max_length) {
    const int in_offset = base_offset + s * chunked_head_size + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = in_buffer[in_offset];
  }

  __syncthreads();

  const int tidx = threadIdx.x + threadIdx.y * chunked_head_size;
  const int tidx_x = tidx % ReorderPastStateParams::kSeqTileSize;
  const int tidx_y = tidx / ReorderPastStateParams::kSeqTileSize;

  const int s2 = s_base + tidx_x;

  if (s2 < max_length) {
    const int out_offset = base_offset + tidx_y * max_length + s2;
    out_buffer[out_offset] = tile[tidx_x][tidx_y];
  }
}

void ReorderPastStatesKernelLauncher(void* out_buffer,
                                     const void* in_buffer,
                                     int batch_size,
                                     int num_heads,
                                     int max_length,
                                     int head_size,
                                     int chunk_size,
                                     hipStream_t stream) {
  // [B, N, max_length, H2(head_size/chunk_size), equv_chunk_size] -> [B, N, H2(head_size/chunk_size), max_length, equv_chunk_size]
  const int chunked_head_size = head_size / chunk_size;
  const dim3 block(chunked_head_size, ReorderPastStateParams::kSeqTileSize);
  const dim3 grid((max_length + ReorderPastStateParams::kSeqTileSize - 1) / ReorderPastStateParams::kSeqTileSize, num_heads, batch_size);
  if (chunk_size == 4 || chunk_size == 8) {
    ReorderPastStatesKernel<<<grid, block, 0, stream>>>(reinterpret_cast<float4*>(out_buffer),
                                                        reinterpret_cast<const float4*>(in_buffer),
                                                        batch_size,
                                                        num_heads,
                                                        max_length,
                                                        chunked_head_size);
  }
}

__global__ void UpdateCacheIndirectionKernel(int32_t* tgt_indir_cache,
                                             const int32_t* src_indir_cache,
                                             const int32_t* beam_ids,
                                             int batch_size,
                                             int beam_width,
                                             int input_seq_length,
                                             int max_seq_length,
                                             int current_length) {
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;
  int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
  const int batch_id = bb_id / beam_width;
  const int beam_id = bb_id % beam_width;

  if (bb_id >= beam_width * batch_size || time_step >= current_length) {
    return;
  }

  const int src_beam = beam_ids[batch_id * beam_width + beam_id] % beam_width;

  const int tgt_offset = batch_id * beam_width * max_seq_length + beam_id * max_seq_length + time_step;

  if (time_step < input_seq_length) {
    // For time steps that correspond to the input sequence,
    // the beam that it comes from is always 0.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(0);
  } else if (time_step == (current_length - 1)) {
    // For the final (newly generated) time step,
    // the beam that it comes from is always the beam that we
    // are currently processing (i.e.) from this point on, these time-steps
    // form the new beams.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(beam_id);
  } else {
    // For all other time-steps, we look up the source indirection, to
    // see which beam it came from based on the `src_beam`.
    const int src_offset = batch_id * beam_width * max_seq_length + src_beam * max_seq_length + time_step;
    tgt_indir_cache[tgt_offset] = src_indir_cache[src_offset];
  }
}

void UpdateCacheIndirectionKernelLauncher(int32_t* tgt_indir_cache,
                                          const int32_t* src_indir_cache,
                                          const int32_t* beam_ids,
                                          int batch_size,
                                          int beam_width,
                                          int input_seq_length,
                                          int max_seq_length,
                                          int current_length,
                                          hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((current_length + block.x - 1) / block.x, batch_size * beam_width);
  UpdateCacheIndirectionKernel<<<grid, block, 0, stream>>>(tgt_indir_cache,
                                                           src_indir_cache,
                                                           beam_ids,
                                                           batch_size,
                                                           beam_width,
                                                           input_seq_length,
                                                           max_seq_length,
                                                           current_length);
}

template <typename T>
__global__ void CopyCrossQKSingleDecodeStepKernel(T* target,  // shape [batch_beam_size, num_alignment_heads, max_length, frames]
                                                  T** qk_layer_pointers,
                                                  int token_index,
                                                  int num_layers,
                                                  int num_heads,
                                                  const int* alignment_heads,
                                                  int frames,
                                                  int max_length) {
  const int pair = blockIdx.x;
  const int num_alignment_heads = gridDim.x;
  const int bbm = blockIdx.y;
  alignment_heads += (pair * 2);
  const int layer = *alignment_heads;
  const int head = *(alignment_heads + 1);

  target += ((int64_t)bbm * num_alignment_heads + pair) * max_length * frames + ((int64_t)token_index * frames);
  T* src = qk_layer_pointers[layer] + ((int64_t)bbm * num_heads + head) * frames;

  for (int tid = threadIdx.x; tid < frames; tid += blockDim.x) {
    target[tid] = src[tid];  // use vectorized read write in future if needed
  }
}

template <typename T>
void LaunchCopyCrossQKSingleDecodeStep(hipStream_t stream,
                                       T* cross_qk_buffer_data,
                                       T** qk_layer_pointers,
                                       int token_index,
                                       int batch_beam_size,
                                       int num_layers,
                                       int num_heads,
                                       int num_alignment_heads,
                                       const int* alignment_heads,
                                       int frames,
                                       int max_length) {
  dim3 block(512);
  dim3 grid(num_alignment_heads, batch_beam_size);

  CopyCrossQKSingleDecodeStepKernel<<<grid, block, 0, stream>>>(cross_qk_buffer_data,
                                                                qk_layer_pointers,
                                                                token_index,
                                                                num_layers,
                                                                num_heads,
                                                                alignment_heads,
                                                                frames,
                                                                max_length);
}

template void LaunchCopyCrossQKSingleDecodeStep(hipStream_t stream,
                                                float* cross_qk_buffer_data,
                                                float** qk_layer_pointers,
                                                int token_index,
                                                int batch_beam_size,
                                                int num_layers,
                                                int num_heads,
                                                int num_alignment_heads,
                                                const int* alignment_heads,
                                                int frames,
                                                int max_length);

template <typename T>
__global__ void CopyDecoderCrossQKAllStepsKernel(int context_decoding_len,
                                                 int num_beams,
                                                 int num_return_sequences,
                                                 int max_length,
                                                 int frames_of_k,
                                                 const T* cross_qk_buffer_data,  // [batch, num_beams, num_alignment_heads, max_length, frames]
                                                 T* cross_qk_output,             // [batch, num_return_sequences, num_alignment_heads, total_decoding_length, frames]
                                                 const int* cache_indir_data) {  // [batch, num_beams, max_length]
  const int pair = blockIdx.y;
  const int num_alignment_heads = gridDim.y;
  const int total_decoding_length = gridDim.x;
  const int token_decoding_index = blockIdx.x;
  const int br = blockIdx.z;
  const int batch = br / num_return_sequences;
  const int ret_seq_id = br % num_return_sequences;

  const int64_t offset_in_cache = ((int64_t)batch * num_return_sequences + ret_seq_id) * max_length + token_decoding_index + context_decoding_len;
  int bi_src = batch * num_beams + cache_indir_data[offset_in_cache];

  T* target = cross_qk_output + (((int64_t)br * num_alignment_heads + (int64_t)pair) * total_decoding_length + token_decoding_index) * frames_of_k;
  const T* src = cross_qk_buffer_data + (((int64_t)bi_src * num_alignment_heads + (int64_t)pair) * max_length + token_decoding_index) * frames_of_k;
  for (int tid = threadIdx.x; tid < frames_of_k; tid += blockDim.x) {
    target[tid] = src[tid];  // use vectorized read write in future if needed
  }
}

template <typename T>
void LaunchFinalizeCrossQK(hipStream_t stream,
                           int iteration_number,
                           int context_decoding_len,
                           int batch_size,
                           int num_beams,
                           int max_length,
                           int num_alignment_heads,
                           int frames_of_k,
                           const T* cross_qk_buffer_data,
                           T* cross_qk_output,
                           int num_return_sequences,
                           const int* cache_indir_data) {
  int64_t br = (int64_t)batch_size * num_return_sequences;
  assert(br < 65536L && num_alignment_heads < 65536);

  const int total_decoding_length = iteration_number - 1;
  dim3 block(512);
  dim3 grid(total_decoding_length, num_alignment_heads, (unsigned)br);

  CopyDecoderCrossQKAllStepsKernel<<<grid, block, 0, stream>>>(context_decoding_len,
                                                               num_beams,
                                                               num_return_sequences,
                                                               max_length,
                                                               frames_of_k,
                                                               cross_qk_buffer_data,
                                                               cross_qk_output,
                                                               cache_indir_data);
}

template void LaunchFinalizeCrossQK(hipStream_t stream,
                                    int iteration_number,
                                    int context_decoding_len,
                                    int batch_size,
                                    int num_beams,
                                    int max_length,
                                    int num_alignment_heads,
                                    int frames_of_k,
                                    const float* cross_qk_buffer_data,
                                    float* cross_qk_output,
                                    int num_return_sequences,
                                    const int* cache_indir_data);

}  // namespace cuda
}  // namespace Generators
